#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <time.h>
#define STB_IMAGE_IMPLEMENTATION
#include "C://Users/Jemo/source/repos/Paralelna implementacija/Paralelna implementacija/stb_image/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "C://Users/Jemo/source/repos/Paralelna implementacija/Paralelna implementacija/stb_image/stb_image_write.h"
void getError(hipError_t err) {
	if (err != hipSuccess) {
		printf("Greska");
	}
}

__global__
void grayscale(const unsigned char* input_rgb, unsigned char* input_gray,const int width, const int height) {
	const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
	if (offset >= width * height) return;
	const unsigned int a = (offset << 1) + (offset); //a = offset*3
		input_gray[offset] = (input_rgb[a] + input_rgb[a + 1] + input_rgb[a + 2]) * .33333333333333;
}

__global__
void negative(const unsigned char* input_rgb,unsigned char* output, const int width, const int height) {
	const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
	if (offset >= width * height) return;
	const unsigned int a = (offset << 1) + (offset); // a = offset*3
	output[a] = 255 - input_rgb[a];
	output[a + 1] = 255 - input_rgb[a+1];
	output[a + 2] = 255 - input_rgb[a+2];
}

	

int main(void) { 
	unsigned char i = 1;
	double time = 0.0;
	char naziv[6] = "a.jpg"; 
	naziv[5] = '\0';
	char izlaz[7] = "ao.jpg"; 
	izlaz[6] = '\0';
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	unsigned char *input, *output;
	for (i = 1; i <= 8; ++i) {
		naziv[0] = i + 48;

		int width, height, channels;

		input= stbi_load(naziv, &width, &height, &channels, 0);
		size_t img_size = width * height * channels;
		int gray_channels = channels == 4 ? 2 : 1;
		size_t gray_img_size = width * height * gray_channels;
		unsigned char *gray_img = (unsigned char*)malloc(gray_img_size);
		unsigned char *dev_input_RGB, *dev_input_GRAY, *dev_input_RGB_2;
		
		getError(hipMalloc((void**)&dev_input_RGB, img_size * sizeof(unsigned char)));
		getError(hipMemcpy(dev_input_RGB, input, img_size * sizeof(unsigned char), hipMemcpyHostToDevice));

		getError(hipMalloc((void**)&dev_input_RGB_2, img_size * sizeof(unsigned char)));
		getError(hipMemcpy(dev_input_RGB_2, input, img_size * sizeof(unsigned char), hipMemcpyHostToDevice));

		getError(hipMalloc((void**)&dev_input_GRAY, gray_img_size * sizeof(unsigned char)));
		getError(hipMemcpy(dev_input_GRAY, gray_img, gray_img_size * sizeof(unsigned char), hipMemcpyHostToDevice));



		dim3 blockDims(640, 1, 1);
		dim3 gridDims((unsigned int)ceil((double)(width*height / (blockDims.x))), 1, 1);
		//dim3 blockDims(32, 32, 1);
		//dim3 gridDims(1 + (width / 32), 1 + (height / 32), 1);


		hipEventRecord(start);
		 // pozvati kernel ovdje
		grayscale <<<gridDims, blockDims >>> (dev_input_RGB,dev_input_GRAY, width, height);
		//negative << <gridDims, blockDims >> > (dev_input_RGB, dev_input_RGB_2, width, height);
		//negative << <numBlocks, threadsPerBlock >> > (dev_input_RGB, dev_input_RGB_2, width, height);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float milis = 0;
		hipEventElapsedTime(&milis, start, stop);
		time += milis;


		getError(hipMemcpy(gray_img, dev_input_GRAY, gray_img_size * sizeof(unsigned char), hipMemcpyDeviceToHost));
		//getError(hipMemcpy(input,dev_input_RGB_2, img_size * sizeof(unsigned char), hipMemcpyDeviceToHost));
		getError(hipFree(dev_input_GRAY));
		getError(hipFree(dev_input_RGB));
		//getError(hipFree(dev_input_RGB_2));
		izlaz[0] = naziv[0];
		stbi_write_jpg(izlaz, width, height, 1, gray_img, 100);
		//stbi_write_jpg(izlaz, width, height, 3, input,100);
	}
	printf("ukupno %f ms \n", time);
	return 0; 
}