
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ 
void grayScale(const uchar4* rgbImage, unsigned char* grayscaleImage, const int height, const int width) {
	int x = blockDim.x * blockDim.x + threadIdx.x; // dobijanje x koordinate piksela
	int y = blockDim.y * blockDim.y + threadIdx.y; // dobijanje y koordinate piksela
	if (x >= width || y >= height) return; // pregled da li je vrijednost proslijedjena funkciji unutar slike
	int pixelIndex = width * y + x;
	grayscaleImage[pixelIndex] = 0.299f * rgbImage[pixelIndex].x + 0.587f * rgbImage[pixelIndex].y + 0.114f * rgbImage[pixelIndex].z;
}


int main()
{
	uchar4 *rgbimg=0;
	unsigned char *grayimg=0;
	int width=0, height=0, channels = 0;
	const int BLOCK_SZ = 128;
	const dim3 blockSz(BLOCK_SZ, BLOCK_SZ, 1);
	const dim3 gridSz(width / BLOCK_SZ + 1, height / BLOCK_SZ + 1, 1);
	grayScale <<<gridSz, blockSz >>> (rgbimg, grayimg, height, width);
	printf("Hello world");
    return 0;
}
