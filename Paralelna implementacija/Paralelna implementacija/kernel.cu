#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <time.h>
#define STB_IMAGE_IMPLEMENTATION
#include "C://Users/Jemo/source/repos/Paralelna implementacija/Paralelna implementacija/stb_image/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "C://Users/Jemo/source/repos/Paralelna implementacija/Paralelna implementacija/stb_image/stb_image_write.h"
void getError(hipError_t err) {
	if (err != hipSuccess) {
		printf("Greska");
	}
}

__global__
void filter(unsigned char* input_rgb, unsigned char* input_gray, int width, int height) {
	const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int a = (offset << 1) + (offset << 0);
	if (offset < width*height)
		input_gray[offset] = (input_rgb[a] + input_rgb[a + 1] + input_rgb[a + 2]) * .33333333333333;
}

int main(void) { 
	unsigned char i = 1;
	double time = 0.0;
	char naziv[6] = "a.jpg"; 
	naziv[5] = '\0';
	char izlaz[7] = "ao.jpg"; 
	izlaz[6] = '\0';
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	unsigned char *input, *output;
	for (i = 1; i <= 8; ++i) {
		naziv[0] = i + 48;

		int width, height, channels;

		input= stbi_load(naziv, &width, &height, &channels, 0);
		size_t img_size = width * height * channels;
		int gray_channels = channels == 4 ? 2 : 1;
		size_t gray_img_size = width * height * gray_channels;
		unsigned char *gray_img = (unsigned char*)malloc(gray_img_size);
		unsigned char *dev_input_RGB, *dev_input_GRAY;
		
		getError(hipMalloc((void**)&dev_input_RGB, img_size * sizeof(unsigned char)));
		getError(hipMemcpy(dev_input_RGB, input, img_size * sizeof(unsigned char), hipMemcpyHostToDevice));

		getError(hipMalloc((void**)&dev_input_GRAY, gray_img_size * sizeof(unsigned char)));
		getError(hipMemcpy(dev_input_GRAY, gray_img, gray_img_size * sizeof(unsigned char), hipMemcpyHostToDevice));



		dim3 blockDims(128, 1, 1);
		dim3 gridDims((unsigned int)ceil((double)(width*height * 3 / blockDims.x)), 1, 1);
		
		hipEventRecord(start);
		 // pozvati kernel ovdje
		filter <<<gridDims, blockDims >>> (dev_input_RGB,dev_input_GRAY, width, height);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float milis = 0;
		hipEventElapsedTime(&milis, start, stop);
		time += milis;


		getError(hipMemcpy(gray_img, dev_input_GRAY, gray_img_size * sizeof(unsigned char), hipMemcpyDeviceToHost));
		getError(hipFree(dev_input_GRAY));
		getError(hipFree(dev_input_RGB));
		izlaz[0] = naziv[0];
		stbi_write_jpg(izlaz, width, height, 1, gray_img, 100);
	}
	printf("ukupno %f ms \n", time);
	return 0; 
}